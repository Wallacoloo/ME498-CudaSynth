#include "hip/hip_runtime.h"
#include "kernel.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <string.h> //for memset
#include <assert.h>
#include <stdlib.h> //for atexit
#include <mutex>
#include <thread> //for unique_lock

#include "defines.h"

#define CIRCULAR_BUFFER_LEN MAX_DELAY_EFFECT_LENGTH

namespace kernel {
	// Must define our own generic complex class to run on both CPU and GPU.
	// std::complex does not have __device__  defines, meaning it will only generate host code.
	template <typename F> class ComplexT {
		F _r, _i;
	public:
		__device__ __host__ ComplexT() : _r(0), _i(0) {}
		__device__ __host__ ComplexT(F real, F imag) : _r(real), _i(imag) {}
		__device__ __host__ ComplexT(F angleRad) {
			sincosf(angleRad, &_i, &_r);
		}
		__device__ __host__ F real() const {
			return _r;
		}
		__device__ __host__ F imag() const {
			return _i;
		}
		__device__ __host__ F magSq() const {
			return _r*_r + _i*_i;
		}
		__device__ __host__ F mag() const {
			return sqrtf(magSq());
		}
		__device__ __host__ F phase() const {
			return atan2(_i, _r);
		}
		__device__ __host__ ComplexT inverse() const {
			// return 1.0/(this)
			// 1 / (a+b*i) = (a-b*i) / (a^2+b^2)
			return ComplexT(_r, -_i) / magSq();
		}
		__device__ __host__ ComplexT operator+(const ComplexT &other) const {
			return ComplexT(_r + other._r, _i + other._i);
		}
		__device__ __host__ ComplexT& operator+=(const ComplexT &other) {
			return (*this = (*this + other));
		}
		__device__ __host__ ComplexT operator*(const ComplexT &other) const {
			//(a+bi)(c+di) = ac + adi + bci + bd(-1)
			// = (ac-bd) + (ad+bc)i;
			return ComplexT(_r*other._r - _i*other._i, _r*other._i + _i*other._r);
		}
		__device__ __host__ ComplexT operator*(F other) const {
			return (*this) * ComplexT(other, 0);
		}
		__device__ __host__ ComplexT operator/(const ComplexT &other) const {
			return (*this) * other.inverse();
		}
		__device__ __host__ ComplexT operator/(F other) const {
			return (*this) * (1.f / other);
		}
		__device__ __host__ ComplexT& operator*=(const ComplexT &other) {
			return (*this = (*this * other));
		}
		__device__ __host__ ComplexT& operator*=(F other)  {
			return (*this) *= ComplexT(other, 0);
		}
		__device__ __host__ ComplexT& operator/=(const ComplexT &other) {
			return (*this) *= other.inverse();
		}
		__device__ __host__ ComplexT& operator/=(F other) {
			return (*this) *= (1.f / other);
		}
		__device__ __host__ ComplexT pow(F n) {
			//(r*e^(i*phase))^n = r^n*e^(i*n*phase)
			F newMag = powf(mag(), n);
			F newPhase = phase()*n;
			return ComplexT(newPhase)*newMag;
		}
	};

	// Efficient way to compute successive sine values
	/*class Sinusoidal {
		// The partial has a phase function, phase(t).
		// For constant frequency, phase(t) = w*t.
		// We need varied frequency over time whenever the frequency changes.
		// Thus, dp/dt = w0 + (w1-w0)/T*t, where T is the time over which the frequency should be altered.
		// Write as dp/dt = w0 + kt
		// and so each sample, the phase accumulator should be multiplied by e^i(w0+kt)
		// This *can* be done efficiently.
		// First, evaluate e^iw0 at the start of the block, store as dP/dt
		//   Also evaluate e^ik(1) as d^2P/dt^2. Each sample, multiply dP/dt = dP/dt * d^2P/dt^2
		
		// Use complex float pairs to represent the phase functions
		typedef ComplexT<float> PhaseT;
		PhaseT phase;
		PhaseT phasePrime;
		PhaseT phaseDoublePrime;
	public:
		Sinusoidal() : phase(1, 0) {}
		// transition from start frequency/depth to end frequency/depth over this block
		__device__ __host__ void newFrequencyAndDepth(float startFreq, float endFreq, float startDepth, float endDepth) {
			float wStart = INV_SAMPLE_RATE * startFreq;
			float wEnd = INV_SAMPLE_RATE * endFreq;
			PhaseT phasePrimeStart = PhaseT(wStart);
			PhaseT phasePrimeEnd = PhaseT(wEnd);
			phasePrime = phasePrimeStart;
			// phasePrimeStart * doublePrime^BUFFER_BLOCK_SIZE = phasePrimeEnd
			// (phasePrimeEnd/phasePrimeStart)^(1.0/BUFFER_BLOCK_SIZE) = doublePrime
			// phaseDoublePrime = PhaseT(powf(phasePrimeEnd/phasePrimeStart, 1.0 / BUFFER_BLOCK_SIZE));
			// Note: (a+bi)^n = (r*e^(i*p))^n = r^n*e^(i*n*p)
			phaseDoublePrime = (phasePrimeEnd / phasePrimeStart).pow(1.f / BUFFER_BLOCK_SIZE);

			// we must avoid the division by zero if the current depth is 0.
			// to avoid this, we just prevent the desired depth from ever *being* zero. 
			// In this way, we also don't lose track of position when the depth is toggled to 0
			//   at the cost of some small inaccuracies under specific conditions
			startDepth = max(0.0001, startDepth);
			endDepth = max(0.0001, endDepth);
			// We cannot transition from startDepth to endDepth linearly,
			//   instead we multiply phase by some scalar each frame.
			// so, mag(frame) = startDepth * k^frame
			// and mag(b=BUFFER_BLOCK_SIZE) = endDepth
			// endDepth = startDepth *k^b
			// (endDepth/startDepth)^(1/b) = k
			float k = powf(endDepth / startDepth, 1.f / BUFFER_BLOCK_SIZE);
			// make current phase magnitude equal to startDepth
			float mag = phase.mag();
			float factor = startDepth / mag;
			phase *= factor;
			phasePrime *= k;
		}
		__device__ __host__ PhaseT next() {
			phasePrime *= phaseDoublePrime;
			phase *= phasePrime;
			return phase;
		}
	};*/

	class Sinusoidal {
		// y(t) = mag(t)*sin(phase(t)), all t in frame offset from block start
		// magnitude of sinusoid
		// mag(t) = mag_c0 + t*mag_c1
		float mag_c0;
		float mag_c1;
		// phase function coefficients:
		// phase(t) = phase_c0 + phase_c1*t + phase_c2*t^2
		float phase_c0, phase_c1, phase_c2;
	public:
		Sinusoidal() : mag_c0(0), mag_c1(0), phase_c0(0), phase_c1(0), phase_c2(0) {}
		// startFreq, endFreq given in rad/sec
		__device__ __host__ void newFrequencyAndDepth(float startFreq, float endFreq, float startDepth, float endDepth) {
			// compute phase function coefficients
			// first, carry over the phase from the end of the previous buffer.
			phase_c0 = phaseAtIdx(BUFFER_BLOCK_SIZE);
			// initial slope is w0
			phase_c1 = startFreq*INV_SAMPLE_RATE;
			float endW = endFreq*INV_SAMPLE_RATE;
			// phase'(BUFFER_BLOCK_SIZE) = endW
			// phase_c1 + 2*t*phase_c2 = endW
			// phase_c2 = (endW - phase_c1) / (2*BUFFER_BLOCK_SIZE)
			phase_c2 = (endW - phase_c1) / (2 * BUFFER_BLOCK_SIZE);
			// compute magnitude function coefficients
			mag_c0 = startDepth;
			float deltaDepth = endDepth - startDepth;
			mag_c1 = deltaDepth * INV_BUFFER_BLOCK_SIZE;
			
		}
		__device__ __host__ float phaseAtIdx(unsigned idx) const {
			return phase_c0 + idx*(phase_c1 + idx*phase_c2);
		}
		__device__ __host__ float magAtIdx(unsigned idx) const {
			return mag_c0 + idx*mag_c1;
		}
		__device__ __host__ float valueAtIdx(unsigned idx) const {
			return magAtIdx(idx)*sinf(phaseAtIdx(idx));
		}
	};

	class ADSRState {
		// mode at start of block
		ADSR::Mode mode;
		// break each mode into a line
		// during the attack/decay mode, 
		//   the block may be up to 2 lines during the block. During sustain/release, just one line.
		// actually, for sufficiently short attack/decay, the note may transition from attack->decay->sustain in one single block
		// The best way to handle this is to define the function like:
		// value(t) = (t <= toggleTime)*(line0_c0+line0_c1*t) + !(t <= toggleTime)*(line1_c0+line1_c1*t)
		// toggleIdx must be signed for an easier release mode implementation
		int toggleIdx;
		// segment coefficients
		float line0_c0, line0_c1, line1_c0, line1_c1;
		__device__ __host__ ADSR::Mode nextMode(ADSR::Mode m) const {
			return (ADSR::Mode)((unsigned)m + 1);
		}
	public:
		// initialized at the start of a note
		ADSRState() : mode(ADSR::AttackMode), toggleIdx(BUFFER_BLOCK_SIZE), line0_c0(0), line0_c1(0), line1_c0(0), line1_c1(0) {}
		__device__ __host__ void atBlockStart(ADSR *start, ADSR *end, unsigned partialIdx, bool released) {
			// get the last value from the previous buffer & increment the mode if needed.
			line0_c0 = valueAtIdx(BUFFER_BLOCK_SIZE);
			// increment the mode if we passed the toggleIdx last buffer
			mode = (ADSR::Mode)((unsigned)mode + ((unsigned)mode < (unsigned)ADSR::EndMode && segmentFromIdx(BUFFER_BLOCK_SIZE) == 1));
			// shift toggleIdx - necessary if in release mode, else has no effect.
			toggleIdx -= BUFFER_BLOCK_SIZE;
			if (released && (unsigned)mode < (unsigned)ADSR::ReleaseMode) {
				// change to release mode if the note was released and is in either the attack, decay or sustain mode
				// release must be handled explicitly because the slope depends on the current value obtained.
				mode = ADSR::ReleaseMode;
				float releaseTime = start->getSegmentLength(ADSR::ReleaseMode, partialIdx);
				float endLevel = start->getSegmentStartLevel(ADSR::EndMode);
				float deltaY = endLevel - line0_c0;
				line0_c1 = deltaY / (releaseTime*SAMPLE_RATE);
				line1_c0 = endLevel;
				line1_c1 = 0;
				toggleIdx = releaseTime*SAMPLE_RATE;
			} else if (mode != ADSR::ReleaseMode) {
				// TODO: segments are not linear; they are quadratic
				float startSlope0 = start->getSegmentSlope(mode, partialIdx);
				float startTrigLevel = start->getSegmentStartLevel(nextMode(mode));
				line0_c1 = startSlope0*INV_SAMPLE_RATE;
				float startSlope1 = start->getSegmentSlope(nextMode(mode), partialIdx);
				line1_c0 = startTrigLevel;
				line1_c1 = startSlope1*INV_SAMPLE_RATE;
				// line0(toggleIdx) == line1(toggleIdx)
				// line0_c0 + line0_c1*toggleIdx = line1_c0 + line1_c1*(toggleIdx-toggleIdx);
				// line0_c1*toggleIdx = line1_c0 - line0_c0
				toggleIdx = (line1_c0 - line0_c0) / line0_c1;
			}
		}
		__device__ __host__ bool segmentFromIdx(unsigned idx) const {
			return idx > toggleIdx;
		}
		__device__ __host__ bool isActiveAtEndOfBlock() const {
			return ((unsigned)mode + segmentFromIdx(BUFFER_BLOCK_SIZE)) < (unsigned)ADSR::EndMode;
		}
		__device__ __host__ float valueAtIdx(unsigned idx) const {
			// return the first line evaluated at idx if idx < toggleIdx, else evaluate the second line at idx
			bool seg = segmentFromIdx(idx);
			return (!seg)*(line0_c0 + idx*line0_c1) + (seg)*(line1_c0 + (idx-toggleIdx)*line1_c1);
		}
	};

	class LFOState {
		ADSRState freqAdsrState;
		ADSRState depthAdsrState;
		Sinusoidal sinusoid;
	public:
		__device__ __host__ void atBlockStart(LFO *start, LFO *end, unsigned partialIdx, bool released) {
			ADSR *freqAdsrStart =  start->getFreqAdsr();
			ADSR *depthAdsrStart = start->getDepthAdsr();
			ADSR *freqAdsrEnd =    end->getFreqAdsr();
			ADSR *depthAdsrEnd =   end->getDepthAdsr();
			// update the ADSR states
			freqAdsrState.atBlockStart(freqAdsrStart, freqAdsrEnd, partialIdx, released);
			depthAdsrState.atBlockStart(depthAdsrStart, depthAdsrEnd, partialIdx, released);
			// obtain the starting and ending frequency and depth.
			// We will then just linearly interpolate over the block.
			//float startFreq = freqAdsrState.next();
			//float startDepth = depthAdsrState.next();
			float startFreq = freqAdsrState.valueAtIdx(0);
			float startDepth = depthAdsrState.valueAtIdx(0);
			float endFreq = freqAdsrState.valueAtIdx(BUFFER_BLOCK_SIZE);
			float endDepth = depthAdsrState.valueAtIdx(BUFFER_BLOCK_SIZE);
			sinusoid.newFrequencyAndDepth(startFreq, endFreq, startDepth, endDepth);
			//sinusoid.newFrequencyAndDepth(startFreq, startFreq, startDepth, startDepth);
		}
		__device__ __host__ float valueAtIdx(unsigned idx) const{
			return sinusoid.valueAtIdx(idx);
		}
	};

	class ADSRLFOEnvelopeState {
		ADSRState adsr;
		LFOState lfo;
	public:
		__device__ __host__ void atBlockStart(ADSRLFOEnvelope *envStart, ADSRLFOEnvelope *envEnd, unsigned partialIdx, bool released) {
			adsr.atBlockStart(envStart->getAdsr(), envEnd->getAdsr(), partialIdx, released);
			lfo.atBlockStart(envStart->getLfo(), envEnd->getLfo(), partialIdx, released);
		}
		__device__ __host__ float productAtIdx(unsigned idx) const {
			return adsr.valueAtIdx(idx) * (1 + lfo.valueAtIdx(idx));
		}
		__device__ __host__ float sumAtIdx(unsigned idx) const {
			return adsr.valueAtIdx(idx) + lfo.valueAtIdx(idx);
		}
		__device__ __host__ bool isActiveAtEndOfBlock() const {
			return adsr.isActiveAtEndOfBlock();
		}
	};

	class DetuneEnvelopeState {
		ADSRLFOEnvelopeState adsrLfoState;
	public:
		__device__ __host__ void atBlockStart(DetuneEnvelope *envStart, DetuneEnvelope *envEnd, unsigned partialIdx, bool released) {
			adsrLfoState.atBlockStart(envStart->getAdsrLfo(), envEnd->getAdsrLfo(), partialIdx, released);
		}
		__device__ __host__ float valueAtIdx(unsigned idx) const {
			return adsrLfoState.sumAtIdx(idx);
		}
	};

	// Contains info about the parameter states at ANY sample in the block
	struct FullBlockParameterInfo {
		ParameterStates start;
		ParameterStates end;
	};

	// Contains extra state information relevant to each individual partial
	struct PartialState {
		Sinusoidal sinusoid;
		ADSRLFOEnvelopeState volumeEnvelope;
		ADSRLFOEnvelopeState stereoPanEnvelope;
		DetuneEnvelopeState detuneEnvelope;
		PartialState() {}
		PartialState(struct SynthState *synthState, unsigned voiceNum, unsigned partialIdx) {}
		__device__ __host__ void atBlockStart(struct SynthVoiceState *voiceState, unsigned partialIdx, float fundamentalFreq, bool released);
	};

	struct SynthVoiceState {
		FullBlockParameterInfo parameterInfo;
		PartialState partialStates[NUM_PARTIALS];
		float sampleBuffer[CIRCULAR_BUFFER_LEN*NUM_CH];
	};

	// Packages all the state-related information for the synth in one class to store persistently on the device
	struct SynthState {
		SynthVoiceState voiceStates[MAX_SIMULTANEOUS_SYNTH_NOTES];
	};

	void PartialState::atBlockStart(struct SynthVoiceState *voiceState, unsigned partialIdx, float fundamentalFreq, bool released) {
		ParameterStates *startParams = &voiceState->parameterInfo.start;
		ParameterStates *endParams = &voiceState->parameterInfo.end;
		detuneEnvelope.atBlockStart(&startParams->detuneEnvelope, &endParams->detuneEnvelope, partialIdx, released);
		
		// calculate the start and end frequency for this block
		float baseFreq = (partialIdx + 1)*fundamentalFreq;
		float detuneStart = detuneEnvelope.valueAtIdx(0);
		float detuneEnd = detuneEnvelope.valueAtIdx(BUFFER_BLOCK_SIZE);
		//float detuneStart = 0.f;
		//float detuneEnd = 0.f;

		// configure the sinusoid to transition from the starting frequency to the end frequency
		sinusoid.newFrequencyAndDepth(baseFreq*(1.f+detuneStart), baseFreq*(1.f+detuneEnd), 1.f, 1.f);
		volumeEnvelope.atBlockStart(&startParams->volumeEnvelope, &endParams->volumeEnvelope, partialIdx, released);
		stereoPanEnvelope.atBlockStart(&startParams->stereoPanEnvelope, &endParams->stereoPanEnvelope, partialIdx, released);
	}

	// this is a circular buffer of sample data (interleaved by channel number) stored on the device
	// It is persistent and lengthy, in order to accomodate the delay effect.
	SynthState *d_synthState = NULL;

	// When running on the cpu, we need to control concurrent access to the synth state
	std::mutex synthStateMutex;

	static void printCudaDeviveProperties(hipDeviceProp_t devProp) {
		// utility function to log device info. Source: https://www.cac.cornell.edu/vw/gpu/example_submit.aspx
		printf("Major revision number:         %d\n", devProp.major);
		printf("Minor revision number:         %d\n", devProp.minor);
		printf("Name:                          %s\n", devProp.name);
		printf("Total global memory:           %lu\n", devProp.totalGlobalMem);
		printf("Total shared memory per block: %lu\n", devProp.sharedMemPerBlock);
		printf("Total registers per block:     %d\n", devProp.regsPerBlock);
		printf("Warp size:                     %d\n", devProp.warpSize);
		printf("Maximum memory pitch:          %lu\n", devProp.memPitch);
		printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
		for (int i = 0; i < 3; ++i) {
			printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
		}
		for (int i = 0; i < 3; ++i) {
			printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
		}
		printf("Clock rate:                    %d\n", devProp.clockRate);
		printf("Total constant memory:         %lu\n", devProp.totalConstMem);
		printf("Texture alignment:             %lu\n", devProp.textureAlignment);
		printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
		printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
		printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	}


	static void checkCudaError(hipError_t e) {
		if (e != hipSuccess) {
			printf("Cuda Error: %s\n", hipGetErrorString(e));
			printf("Aborting\n");
			exit(1);
		}
	}

	static bool _hasCudaDevice() {
		int deviceCount;
		hipError_t err = hipGetDeviceCount(&deviceCount);
		// if we get a cuda error, it may be because the system has no cuda dlls.
		bool useCuda = (err == hipSuccess && deviceCount != 0) && !NEVER_USE_CUDA;
		printf("Using Cuda? %i\n", useCuda);
		if (useCuda) {
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, 0);
			printCudaDeviveProperties(prop);
		}
		return useCuda;
	}

	static bool hasCudaDevice() {
		//only check for the presence of a device once.
		static bool hasDevice = _hasCudaDevice();
		return hasDevice;
	}

	// code to run at shutdown (free buffers, etc)
	static void teardown() {
		std::unique_lock<std::mutex> stateLock(synthStateMutex);
		// free the sample buffer if we allocated it and it hasn't already been freed.
		if (d_synthState != NULL) {
			if (hasCudaDevice()) {
				checkCudaError(hipFree(d_synthState));
			} else {
				free(d_synthState);
			}
			// avoid double-frees
			d_synthState = NULL;
		}
	}

	// code to run on first-time audio calculation
	static void startup() {
		atexit(&teardown);
		//SynthState defaultState;
		std::unique_lock<std::mutex> stateLock(synthStateMutex);
		if (hasCudaDevice()) {
			// allocate sample buffer on device
			checkCudaError(hipMalloc(&d_synthState, sizeof(SynthState)));
			checkCudaError(hipMemset(d_synthState, 0, sizeof(SynthState)));
			//checkCudaError(hipMemcpy(d_synthState, &defaultState, sizeof(SynthState), hipMemcpyHostToDevice));
		} else {
			// allocate sample buffer on cpu
			d_synthState = (SynthState*)malloc(sizeof(SynthState));
			memset(d_synthState, 0, sizeof(SynthState));
			//memcpy(d_synthState, &defaultState, sizeof(SynthState));
		}
	}

	static void doStartupOnce() {
		static bool hasInit = false;
		if (!hasInit) {
			startup();
			hasInit = true;
		}
	}

	// called for each partial to sum their outputs together.
	__device__ __host__ void reduceOutputs(SynthVoiceState *voiceState, unsigned partialIdx, int sampleIdx, float outputL, float outputR) {
		//algorithm: given 8 outputs, [0, 1, 2, 3, 4, 5, 6, 7]
		//first iteration: 4 active threads. 
		//  Thread 0 adds i0 to i(0+4). Thread 1 adds i1 to i(1+4). Thread 2 adds i2 to i(2+4). Thread 3 adds i3 to i(3+4)
		//  Output now: [4, 6, 8, 10,   4, 5, 6, 7]
		//second iteration: 2 active threads.
		//  Thread 0 adds i0 to i(0+2). Thread 1 adds i1 to i(1+2)
		//  Output now: [12, 16,   8, 10, 4, 5, 6, 7]
		//third iteration: 1 active thread.
		//  Thread 0 adds i0 to i(0+1).
		//  Output now: [28,   16, 8, 10, 4, 5, 6, 7]
		//fourth iteration: 0 active threads -> exit
		unsigned bufferIdx = NUM_CH * (sampleIdx % CIRCULAR_BUFFER_LEN);
#ifdef __CUDA_ARCH__
		//device code
		// This reduction method requires a temporary array in shared memory.
		__shared__ float partialReductionOutputs[NUM_PARTIALS*NUM_CH];

		partialReductionOutputs[NUM_CH*partialIdx + 0] = outputL;
		partialReductionOutputs[NUM_CH*partialIdx + 1] = outputR;
		unsigned numActiveThreads = NUM_PARTIALS / 2;
		while (numActiveThreads > 0) {
			__syncthreads();
			if (partialIdx < numActiveThreads) {
				partialReductionOutputs[NUM_CH*partialIdx + 0] += partialReductionOutputs[NUM_CH*partialIdx + numActiveThreads*NUM_CH + 0];
				partialReductionOutputs[NUM_CH*partialIdx + 1] += partialReductionOutputs[NUM_CH*partialIdx + numActiveThreads*NUM_CH + 1];
			}
			numActiveThreads /= 2;
		}
		if (partialIdx == 0) {
			voiceState->sampleBuffer[bufferIdx + 0] = partialReductionOutputs[0];
			voiceState->sampleBuffer[bufferIdx + 1] = partialReductionOutputs[1];
		}
#else
		//host code
		//Since everything's computed iteratively, we can just add our outputs directly to the buffer.
		//First write to this sample must zero-initialize the buffer (not required in the GPU code).
		if (partialIdx == 0) {
			voiceState->sampleBuffer[bufferIdx + 0] = 0;
			voiceState->sampleBuffer[bufferIdx + 1] = 0;
		}
		voiceState->sampleBuffer[bufferIdx + 0] += outputL;
		voiceState->sampleBuffer[bufferIdx + 1] += outputR;
#endif
	}

	// called at the end of the block.
	// if parameterInfo.start != parameterInfo.end, then we copy the end parameters of this block to the start parameters for the next block.
	// this needs to be called for each sine wave.
	__device__ __host__ void updateVoiceParametersIfNeeded(SynthVoiceState *voiceState, unsigned voiceNum, unsigned partialIdx) {
		/*int transferSize = 16;
		int totalBytesToCopy = sizeof(ParameterStates);
		int numTransfers = (totalBytesToCopy + transferSize - 1) / transferSize;
		int numTransfersPerThread = (numTransfers + NUM_PARTIALS - 1) / NUM_PARTIALS;*/
		if (partialIdx == NUM_PARTIALS-1) {
			// TODO: only do this copy if the parameters have changed
			memcpy(&voiceState->parameterInfo.start, &voiceState->parameterInfo.end, sizeof(ParameterStates));
		}

	}

	// compute the output for ONE sine wave over the current block
	__device__ __host__ void computePartialOutput(SynthState *synthState, unsigned voiceNum, unsigned baseIdx, unsigned partialIdx, float fundamentalFreq, bool released) {
		SynthVoiceState *voiceState = &synthState->voiceStates[voiceNum];
		PartialState* myState = &voiceState->partialStates[partialIdx];
		myState->atBlockStart(voiceState, partialIdx, fundamentalFreq, released);
		// Get the base partial level (the hand-drawn frequency weights)
		float level = (1.0 / NUM_PARTIALS) * voiceState->parameterInfo.start.partialLevels[partialIdx];
		//printf("partialIdx: %i\n", partialIdx);
		for (int sampleIdx = 0; sampleIdx < BUFFER_BLOCK_SIZE; ++sampleIdx) {
			// Extract the sinusoidal portion of the wave.
			// float sinusoid = myState->sinusoid.next().imag();
			float sinusoid = myState->sinusoid.valueAtIdx(sampleIdx);
			// Get the ADSR/LFO volume envelope
			//float envelope = myState->volumeEnvelope.nextAsProduct();
			//float pan = myState->stereoPanEnvelope.nextAsSum();
			float envelope = myState->volumeEnvelope.productAtIdx(sampleIdx);
			float pan = myState->stereoPanEnvelope.sumAtIdx(sampleIdx);
			float unpanned = level*envelope*sinusoid;
			//float outputL = unpanned;
			//float outputR = unpanned;
			// full left = -1 pan. full right = +1 pan.
			// Use circular panning, where L^2 + R^2 = 1.0
			//   R(+1.0 pan) = 1.0, L(-1.0 pan) = 0.0, R(0.0 pan) = sqrt(1/2)
			//   L(+1.0 pan) = 0.0, L(-1.0 pan) = 1.0, L(0.0 pan) = sqrt(1/2)
			//   then R(pan) = sqrt((1+pan)/2)
			//   L(pan) = sqrt((1-pan)/2)
			// Note that L(pan)^2 + R(pan)^2 = 1.0, so energy is constant.
			// Must deal with pan values of magnitude > 1.0
			// Note the analog between sinusoidals:
			// sin(x)^2 + cos(x)^2 = 1.0 = L(pan)^2 + R(pan)^2
			// sin(pi/4) = cos(pi/4) = L(0.0) = R(0.0) = sqrt(1/2)
			// cos(0.0) = L(-1.0) = 1.0
			// cos(pi/2) = L(1.0) = 0.0
			// sin(0.0) = R(-1.0) = 0.0
			// sin(pi/2) = R(1.0) = 1.0
			// So, L(pan) = cos(pi/4 + pi/4*pan) = cos(pi/4*(1+pan))
			//     R(pan) = sin(pi/4 + pi/4*pan) = sin(pi/4*(1+pan))
			float angle = PI / 4 * (1 + pan);
			float outputL = unpanned * cosf(angle);
			float outputR = unpanned * sinf(angle);
			//float outputL = unpanned * sqrt(0.5*(1-pan));
			//float outputR = unpanned * sqrt(0.5*(1+pan));
			//linear pan implementation:
			//float outputL = unpanned * 0.5*(1 - pan);
			//float outputR = unpanned * 0.5*(1 + pan);

			reduceOutputs(voiceState, partialIdx, baseIdx + sampleIdx, outputL, outputR);
		}
		updateVoiceParametersIfNeeded(voiceState, voiceNum, partialIdx);
		// TODO: use a proper reduction algorithm to determine when the note is complete
		if (partialIdx == NUM_PARTIALS-1 && !myState->volumeEnvelope.isActiveAtEndOfBlock()) {
			// signal no more samples
			unsigned bufferEndIdx = NUM_CH * ((baseIdx + (BUFFER_BLOCK_SIZE - 1)) % CIRCULAR_BUFFER_LEN);
			voiceState->sampleBuffer[bufferEndIdx] = NAN;
		}
	}

	__global__ void evaluateSynthVoiceBlockKernel(SynthState *synthState, unsigned voiceNum, unsigned baseIdx, float fundamentalFreq, bool released) {
		int partialNum = threadIdx.x;
		computePartialOutput(synthState, voiceNum, baseIdx, partialNum, fundamentalFreq, released);
	}

	__host__ void evaluateSynthVoiceBlockOnCpu(float bufferB[BUFFER_BLOCK_SIZE*NUM_CH], unsigned voiceNum, unsigned sampleIdx, float fundamentalFreq, bool released) {
		// need to obtain a lock on the synth state
		std::unique_lock<std::mutex> stateLock(synthStateMutex);
		// move pointer to d_synthState into a local for easy debugging
		SynthState *synthState = d_synthState;
		for (int partialIdx = 0; partialIdx < NUM_PARTIALS; ++partialIdx) {
			computePartialOutput(synthState, voiceNum, sampleIdx, partialIdx, fundamentalFreq, released);
		}
		unsigned bufferStartIdx = NUM_CH * (sampleIdx % CIRCULAR_BUFFER_LEN);
		memcpy(bufferB, &synthState->voiceStates[voiceNum].sampleBuffer[bufferStartIdx], BUFFER_BLOCK_SIZE*NUM_CH*sizeof(float));
	}

	__host__ void evaluateSynthVoiceBlockCuda(float bufferB[BUFFER_BLOCK_SIZE*NUM_CH], unsigned voiceNum, unsigned sampleIdx, float fundamentalFreq, bool released) {
		evaluateSynthVoiceBlockKernel << <1, NUM_PARTIALS >> >(d_synthState, voiceNum, sampleIdx, fundamentalFreq, released);

		checkCudaError(hipGetLastError()); //check if error in kernel launch
		checkCudaError(hipDeviceSynchronize()); //check for error INSIDE the kernel

		//copy memory into the cpu buffer
		//Note: this will wait for the kernel to complete first.
		unsigned bufferStartIdx = NUM_CH * (sampleIdx % CIRCULAR_BUFFER_LEN);
		checkCudaError(hipMemcpy(bufferB, &d_synthState->voiceStates[voiceNum].sampleBuffer[bufferStartIdx], BUFFER_BLOCK_SIZE*NUM_CH*sizeof(float), hipMemcpyDeviceToHost));
	}

	void evaluateSynthVoiceBlock(float *bufferB, unsigned voiceNum, unsigned baseIdx, float fundamentalFreq, bool released) {
		doStartupOnce();
		if (hasCudaDevice()) {
			evaluateSynthVoiceBlockCuda(bufferB, voiceNum, baseIdx, fundamentalFreq, released);
		} else {
			evaluateSynthVoiceBlockOnCpu(bufferB, voiceNum, baseIdx, fundamentalFreq, released);
		}
	}

	static void memcpyHostToSynthState(void *dest, const void *src, std::size_t numBytes) {
		// if running on device, copy params to GPU via hipMemcpy, else normal memcpy on cpu.
		if (hasCudaDevice()) {
			// hipMemcpy is synchronous, so concurrency is dealt with automatically
			checkCudaError(hipMemcpy(dest, src, numBytes, hipMemcpyHostToDevice));
		} else {
			// else, copy them using normal memcpy
			// Must first obtain a lock to the synth data.
			std::unique_lock<std::mutex> stateLock(synthStateMutex);
			memcpy(dest, src, numBytes);
		}
	}

	static void memsetSynthState(void *dest, int value, std::size_t numBytes) {
		// if running on device, use hipMemset, else normal memset
		if (hasCudaDevice()) {
			// hipMemset is synchronous, so concurrency is dealt with automatically
			checkCudaError(hipMemset(dest, value, numBytes));
		} else {
			// else, use normal memset
			// Must first obtain a lock to the synth data.
			std::unique_lock<std::mutex> stateLock(synthStateMutex);
			memset(dest, value, numBytes);
		}
	}

	static void copyParameterStates(const ParameterStates *newParameters, ParameterStates *dest) {
		memcpyHostToSynthState(dest, newParameters, sizeof(ParameterStates));
	}

	void parameterStatesChanged(const ParameterStates *newParameters) {
		doStartupOnce();
		static bool hasInitStartParams = false;

		for (int i = 0; i < MAX_SIMULTANEOUS_SYNTH_NOTES; ++i) {
			// If this is the first time we've received parameter states, then that means parameterInfo.start is uninitialized.
			if (!hasInitStartParams) {
				copyParameterStates(newParameters, &d_synthState->voiceStates[i].parameterInfo.start);
			}
			copyParameterStates(newParameters, &d_synthState->voiceStates[i].parameterInfo.end);
		}
		hasInitStartParams = true;
	}

	void onNoteStart(unsigned voiceNum) {
		doStartupOnce();
		// need to go through and properly initialize all the note's state information:
		//   partial phases, ADSR states, etc.
		PartialState partialStates[NUM_PARTIALS];
		for (int i = 0; i < NUM_PARTIALS; ++i) {
			partialStates[i] = PartialState(d_synthState, voiceNum, i);
		}
		memcpyHostToSynthState(&d_synthState->voiceStates[voiceNum].partialStates, partialStates, sizeof(PartialState)*NUM_PARTIALS);
		memsetSynthState(&d_synthState->voiceStates[voiceNum].sampleBuffer, 0, CIRCULAR_BUFFER_LEN*NUM_CH*sizeof(float));
	}

}