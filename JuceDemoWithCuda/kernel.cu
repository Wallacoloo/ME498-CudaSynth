#include "hip/hip_runtime.h"
#include "kernel.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <string.h> //for memset
#include <assert.h>
#include <stdlib.h> //for atexit
#include <mutex>
#include <thread> //for unique_lock

#include "defines.h"

#define CIRCULAR_BUFFER_LEN MAX_DELAY_EFFECT_LENGTH

namespace kernel {
	class Sinusoidal {
		// y(t) = mag(t)*sin(phase(t)), all t in frame offset from block start
		// magnitude of sinusoid
		// mag(t) = mag_c0 + t*mag_c1
		float mag_c0;
		float mag_c1;
		// phase function coefficients:
		// phase(t) = phase_c0 + phase_c1*t + phase_c2*t^2
		float phase_c0, phase_c1, phase_c2;
		__device__ __host__ float phaseAtIdx(unsigned idx) const {
			return phase_c0 + idx*(phase_c1 + idx*phase_c2);
		}
		__device__ __host__ float magAtIdx(unsigned idx) const {
			return mag_c0 + idx*mag_c1;
		}
	public:
		Sinusoidal() : mag_c0(0), mag_c1(0), phase_c0(0), phase_c1(0), phase_c2(0) {}
		// startFreq, endFreq given in rad/sec
		__device__ __host__ void newFrequencyAndDepth(float startFreq, float endFreq, float startDepth, float endDepth) {
			// compute phase function coefficients
			// first, carry over the phase from the end of the previous buffer.
			phase_c0 = phaseAtIdx(BUFFER_BLOCK_SIZE);
			// initial slope is w0
			phase_c1 = startFreq*INV_SAMPLE_RATE;
			float endW = endFreq*INV_SAMPLE_RATE;
			// phase'(BUFFER_BLOCK_SIZE) = endW
			// phase_c1 + 2*t*phase_c2 = endW
			// phase_c2 = (endW - phase_c1) / (2*BUFFER_BLOCK_SIZE)
			phase_c2 = (endW - phase_c1) / (2 * BUFFER_BLOCK_SIZE);
			// compute magnitude function coefficients
			mag_c0 = startDepth;
			float deltaDepth = endDepth - startDepth;
			mag_c1 = deltaDepth * INV_BUFFER_BLOCK_SIZE;
			
		}
		__device__ __host__ float valueAtIdx(unsigned idx) const {
			return magAtIdx(idx)*sinf(phaseAtIdx(idx));
		}
	};

	class ADSRState {
		// better approach (not yet implemented):
		//   upon ADSR change:
		//     determine the current segment, current length, current value and current time (current time MUST be stored)
		//     determine the new end value and new length
		//     alter coefficients such that current values match and so that the end value will be reached at the new length.
		// proportion of way through ADSR envelope at start;
		// 0 <= P < 1 for attack phase,
		// 1 <= P < 2 for decay phase,
		// 2 <= P < 3 for sustain phase,
		// 3 <= P < 4 for release phase,
		// 4 <= P indicates note end.
		float P;
		// break each mode into a line
		// during the attack/decay mode, 
		//   the block may be up to 2 lines during the block. During sustain/release, just one line.
		// actually, for sufficiently short attack/decay, the note may transition from attack->decay->sustain in one single block
		// in this case, it is sufficient to clamp the index to the point at which we switch from decay to sustain mode, since decay(last) == sustain
		// The best way to handle this is to define the function like:
		// value(t) = (t <= toggleTime)*(line0_c0+line0_c1*t+line0_c2*t^2) + !(t <= toggleTime)*(line1_c0+line1_c1*t+line1_c2*t^2)
		// any index > clampIdx should return the same value as clampIdx. This is for handling 3-part envelopes where the final portion is constant.
		float clampIdx;
		// segment coefficients
		// the values these take are in the same units as 'P'
		float line0_c0, line0_c1, line1_c0, line1_c1;
		float line0_invLength;
		float line1_invLength;
		__device__ __host__ ADSR::Mode getMode() const {
			return (ADSR::Mode)(unsigned)P;
		}
		__device__ __host__ ADSR::Mode nextMode(ADSR::Mode m) const {
			return (ADSR::Mode)((unsigned)m + 1);
		}
		__device__ __host__ float pFromIdx(float idx) const {
			return P + idx*line0_invLength;
		}
		__device__ __host__ bool segmentFromP(float pIdx) const {
			return pIdx >= (unsigned)nextMode(getMode());
		}
		__device__ __host__ float interpolate(float position, float a, float b) const {
			// construct a function where f(0) = a, f(1) = b, and return f(position)
			return a + (b - a)*position;
		}
	public:
		// initialized at the start of a note
		ADSRState() : P(0), clampIdx(BUFFER_BLOCK_SIZE), 
			line0_c0(0), line0_c1(0), 
			line1_c0(0), line1_c1(0),
			line0_invLength(1e-7f), line1_invLength(1e-7f) {}
		__device__ __host__ void atBlockStart(ADSR *start, ADSR *end, unsigned partialIdx, bool released) {
			// preserve previous value
			float prevValue = valueAtIdx(BUFFER_BLOCK_SIZE);
			// track position in envelope
			float idxOfSwitch = ((unsigned)nextMode(getMode()) - P) / line0_invLength;
			idxOfSwitch = min(idxOfSwitch, (float)BUFFER_BLOCK_SIZE);
			// add accumulated index change from each segment
			P += idxOfSwitch*line0_invLength + (min(clampIdx, (float)BUFFER_BLOCK_SIZE) - idxOfSwitch)*line1_invLength;
			// if we're released, skip to release mode (or further)
			P = max(P, released*(float)(unsigned)ADSR::ReleaseMode);
			// update slope of segment and rate at which we progress:
			float line0_length = end->getSegmentLength(getMode(), partialIdx) * SAMPLE_RATE;
			line0_invLength = 1.f / line0_length;
			float line1_length = end->getSegmentLength(nextMode(getMode()), partialIdx) * SAMPLE_RATE;
			line1_invLength = 1.f / line1_length;
			// calculate endpoint values for our lines
			float line0_relPositionAtBufferBlockSize = pFromIdx(BUFFER_BLOCK_SIZE) - (float)(unsigned)getMode();
			float line0_valueAtBufferBlockSize = interpolate(line0_relPositionAtBufferBlockSize, end->getSegmentStartLevel(getMode()), end->getSegmentStartLevel(nextMode(getMode())));
			float line1_startValue = end->getSegmentStartLevel(nextMode(getMode()));
			// update c0 and c1 based on the following constraints:
			// value(P) == prevValue
			// value(pFromIdx(BUFFER_BLOCK_SIZE)) == line0_valueAtBufferBlockSize
			// c0 + c1*P == prevValue
			// c0 + c1*P2 == endValue
			// c1*(P2-P) == endValue-prevValue -> c1 = (endValue-prevValue)/(P2-P)
			// c0 = prevValue - c1*P;
			// line0_c1 = (line0_valueAtBufferBlockSize - prevValue) / (pFromIdx(BUFFER_BLOCK_SIZE) - P);
			// line0_c1 = (line0_valueAtBufferBlockSize - prevValue) / (BUFFER_BLOCK_SIZE*line0_invLength);
			line0_c1 = (line0_valueAtBufferBlockSize - prevValue) * line0_length * INV_BUFFER_BLOCK_SIZE;
			line0_c0 = prevValue - line0_c1*P;
			// then calculate the coefficients for the second portion of the line
			// line1(endP) == startVal
			// line1(endP+length1*sample_rate*IL0) == endVal
			unsigned endP = (unsigned)nextMode(getMode());
			float line1_endValue = end->getSegmentStartLevel(nextMode(nextMode(getMode())));
			// c0 + c1*endP == startVal
			// c0 + c1*endP + c1*length1*IL0 == endVal
			// c1*length1*IL0 == endVal - startVal
			line1_c1 = (line1_endValue - line1_startValue) / (line1_length*line0_invLength);
			// line1_c0 + line1_c1*endP == startValue
			line1_c0 = line1_startValue - line1_c1*endP;
			// then determine the value for clampIdx
			// endP+length1*IL0 == P+clampIdx*IL0
			// (endP-P)/IL0 + length1 = clampIdx
			float seg1StartIdx = (endP - P) / line0_invLength;
			float seg1EndIdx = seg1StartIdx + line1_length;
			clampIdx = seg1EndIdx;
		}
		__device__ __host__ bool isActiveAtEndOfBlock() const {
			return pFromIdx(BUFFER_BLOCK_SIZE) < (unsigned)ADSR::EndMode;
		}
		__device__ __host__ float valueAtIdx(unsigned idx) const {
			// return the first line evaluated at idx if idx < toggleIdx, else evaluate the second line at idx
			float idxAsFloat = min((float)idx, clampIdx);
			float pIdx = pFromIdx(idxAsFloat);
			bool seg = segmentFromP(pIdx);
			return (!seg)*(line0_c0 + pIdx*line0_c1) + (seg)*(line1_c0 + pIdx*line1_c1);
		}
	};

	class LFOState {
		ADSRState freqAdsrState;
		ADSRState depthAdsrState;
		Sinusoidal sinusoid;
	public:
		__device__ __host__ void atBlockStart(LFO *start, LFO *end, unsigned partialIdx, bool released) {
			ADSR *freqAdsrStart =  start->getFreqAdsr();
			ADSR *depthAdsrStart = start->getDepthAdsr();
			ADSR *freqAdsrEnd =    end->getFreqAdsr();
			ADSR *depthAdsrEnd =   end->getDepthAdsr();
			// update the ADSR states
			freqAdsrState.atBlockStart(freqAdsrStart, freqAdsrEnd, partialIdx, released);
			depthAdsrState.atBlockStart(depthAdsrStart, depthAdsrEnd, partialIdx, released);
			// obtain the starting and ending frequency and depth.
			// We will then just linearly interpolate over the block.
			float startFreq = freqAdsrState.valueAtIdx(0);
			float startDepth = depthAdsrState.valueAtIdx(0);
			float endFreq = freqAdsrState.valueAtIdx(BUFFER_BLOCK_SIZE);
			float endDepth = depthAdsrState.valueAtIdx(BUFFER_BLOCK_SIZE);
			sinusoid.newFrequencyAndDepth(startFreq, endFreq, startDepth, endDepth);
		}
		__device__ __host__ float valueAtIdx(unsigned idx) const{
			return sinusoid.valueAtIdx(idx);
		}
	};

	class ADSRLFOEnvelopeState {
		ADSRState adsr;
		LFOState lfo;
	public:
		__device__ __host__ void atBlockStart(ADSRLFOEnvelope *envStart, ADSRLFOEnvelope *envEnd, unsigned partialIdx, bool released) {
			adsr.atBlockStart(envStart->getAdsr(), envEnd->getAdsr(), partialIdx, released);
			lfo.atBlockStart(envStart->getLfo(), envEnd->getLfo(), partialIdx, released);
		}
		__device__ __host__ float productAtIdx(unsigned idx) const {
			return adsr.valueAtIdx(idx) * (1 + lfo.valueAtIdx(idx));
		}
		__device__ __host__ float sumAtIdx(unsigned idx) const {
			return adsr.valueAtIdx(idx) + lfo.valueAtIdx(idx);
		}
		__device__ __host__ bool isActiveAtEndOfBlock() const {
			return adsr.isActiveAtEndOfBlock();
		}
	};

	class DetuneEnvelopeState {
		ADSRLFOEnvelopeState adsrLfoState;
	public:
		__device__ __host__ void atBlockStart(DetuneEnvelope *envStart, DetuneEnvelope *envEnd, unsigned partialIdx, bool released) {
			adsrLfoState.atBlockStart(envStart->getAdsrLfo(), envEnd->getAdsrLfo(), partialIdx, released);
		}
		__device__ __host__ float valueAtIdx(unsigned idx) const {
			return adsrLfoState.sumAtIdx(idx);
		}
	};

	// Contains info about the parameter states at ANY sample in the block
	struct FullBlockParameterInfo {
		ParameterStates start;
		ParameterStates end;
	};

	// Contains extra state information relevant to each individual partial
	struct PartialState {
		Sinusoidal sinusoid;
		ADSRLFOEnvelopeState volumeEnvelope;
		ADSRLFOEnvelopeState stereoPanEnvelope;
		DetuneEnvelopeState detuneEnvelope;
		PartialState() {}
		PartialState(struct SynthState *synthState, unsigned voiceNum, unsigned partialIdx) {}
		__device__ __host__ void atBlockStart(struct SynthVoiceState *voiceState, unsigned partialIdx, float fundamentalFreq, bool released);
	};

	struct SynthVoiceState {
		FullBlockParameterInfo parameterInfo;
		PartialState partialStates[NUM_PARTIALS];
		float sampleBuffer[CIRCULAR_BUFFER_LEN*NUM_CH];
	};

	// Packages all the state-related information for the synth in one class to store persistently on the device
	struct SynthState {
		SynthVoiceState voiceStates[MAX_SIMULTANEOUS_SYNTH_NOTES];
	};

	void PartialState::atBlockStart(struct SynthVoiceState *voiceState, unsigned partialIdx, float fundamentalFreq, bool released) {
		ParameterStates *startParams = &voiceState->parameterInfo.start;
		ParameterStates *endParams = &voiceState->parameterInfo.end;
		detuneEnvelope.atBlockStart(&startParams->detuneEnvelope, &endParams->detuneEnvelope, partialIdx, released);
		
		// calculate the start and end frequency for this block
		float baseFreq = (partialIdx + 1)*fundamentalFreq;
		float detuneStart = detuneEnvelope.valueAtIdx(0);
		float detuneEnd = detuneEnvelope.valueAtIdx(BUFFER_BLOCK_SIZE);

		// configure the sinusoid to transition from the starting frequency to the end frequency
		sinusoid.newFrequencyAndDepth(baseFreq*(1.f+detuneStart), baseFreq*(1.f+detuneEnd), 1.f, 1.f);
		volumeEnvelope.atBlockStart(&startParams->volumeEnvelope, &endParams->volumeEnvelope, partialIdx, released);
		stereoPanEnvelope.atBlockStart(&startParams->stereoPanEnvelope, &endParams->stereoPanEnvelope, partialIdx, released);
	}

	// this is a circular buffer of sample data (interleaved by channel number) stored on the device
	// It is persistent and lengthy, in order to accomodate the delay effect.
	SynthState *d_synthState = NULL;

	// When running on the cpu, we need to control concurrent access to the synth state
	std::mutex synthStateMutex;

	static void printCudaDeviveProperties(hipDeviceProp_t devProp) {
		// utility function to log device info. Source: https://www.cac.cornell.edu/vw/gpu/example_submit.aspx
		printf("Major revision number:         %d\n", devProp.major);
		printf("Minor revision number:         %d\n", devProp.minor);
		printf("Name:                          %s\n", devProp.name);
		printf("Total global memory:           %lu\n", devProp.totalGlobalMem);
		printf("Total shared memory per block: %lu\n", devProp.sharedMemPerBlock);
		printf("Total registers per block:     %d\n", devProp.regsPerBlock);
		printf("Warp size:                     %d\n", devProp.warpSize);
		printf("Maximum memory pitch:          %lu\n", devProp.memPitch);
		printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
		for (int i = 0; i < 3; ++i) {
			printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
		}
		for (int i = 0; i < 3; ++i) {
			printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
		}
		printf("Clock rate:                    %d\n", devProp.clockRate);
		printf("Total constant memory:         %lu\n", devProp.totalConstMem);
		printf("Texture alignment:             %lu\n", devProp.textureAlignment);
		printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
		printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
		printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	}


	static void checkCudaError(hipError_t e) {
		if (e != hipSuccess) {
			printf("Cuda Error: %s\n", hipGetErrorString(e));
			printf("Aborting\n");
			exit(1);
		}
	}

	static bool _hasCudaDevice() {
		int deviceCount;
		hipError_t err = hipGetDeviceCount(&deviceCount);
		// if we get a cuda error, it may be because the system has no cuda dlls.
		bool useCuda = (err == hipSuccess && deviceCount != 0) && !NEVER_USE_CUDA;
		printf("Using Cuda? %i\n", useCuda);
		if (useCuda) {
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, 0);
			printCudaDeviveProperties(prop);
		}
		return useCuda;
	}

	static bool hasCudaDevice() {
		//only check for the presence of a device once.
		static bool hasDevice = _hasCudaDevice();
		return hasDevice;
	}

	// code to run at shutdown (free buffers, etc)
	static void teardown() {
		std::unique_lock<std::mutex> stateLock(synthStateMutex);
		// free the sample buffer if we allocated it and it hasn't already been freed.
		if (d_synthState != NULL) {
			if (hasCudaDevice()) {
				checkCudaError(hipFree(d_synthState));
			} else {
				free(d_synthState);
			}
			// avoid double-frees
			d_synthState = NULL;
		}
	}

	// code to run on first-time audio calculation
	static void startup() {
		atexit(&teardown);
		//SynthState defaultState;
		std::unique_lock<std::mutex> stateLock(synthStateMutex);
		if (hasCudaDevice()) {
			// allocate sample buffer on device
			checkCudaError(hipMalloc(&d_synthState, sizeof(SynthState)));
			checkCudaError(hipMemset(d_synthState, 0, sizeof(SynthState)));
			//checkCudaError(hipMemcpy(d_synthState, &defaultState, sizeof(SynthState), hipMemcpyHostToDevice));
		} else {
			// allocate sample buffer on cpu
			d_synthState = (SynthState*)malloc(sizeof(SynthState));
			memset(d_synthState, 0, sizeof(SynthState));
			//memcpy(d_synthState, &defaultState, sizeof(SynthState));
		}
	}

	static void doStartupOnce() {
		static bool hasInit = false;
		if (!hasInit) {
			startup();
			hasInit = true;
		}
	}

	// called for each partial to sum their outputs together.
	__device__ __host__ void reduceOutputs(SynthVoiceState *voiceState, unsigned partialIdx, int sampleIdx, float outputL, float outputR) {
		//algorithm: given 8 outputs, [0, 1, 2, 3, 4, 5, 6, 7]
		//first iteration: 4 active threads. 
		//  Thread 0 adds i0 to i(0+4). Thread 1 adds i1 to i(1+4). Thread 2 adds i2 to i(2+4). Thread 3 adds i3 to i(3+4)
		//  Output now: [4, 6, 8, 10,   4, 5, 6, 7]
		//second iteration: 2 active threads.
		//  Thread 0 adds i0 to i(0+2). Thread 1 adds i1 to i(1+2)
		//  Output now: [12, 16,   8, 10, 4, 5, 6, 7]
		//third iteration: 1 active thread.
		//  Thread 0 adds i0 to i(0+1).
		//  Output now: [28,   16, 8, 10, 4, 5, 6, 7]
		//fourth iteration: 0 active threads -> exit
		unsigned bufferIdx = NUM_CH * (sampleIdx % CIRCULAR_BUFFER_LEN);
#ifdef __CUDA_ARCH__
		//device code
		// This reduction method requires a temporary array in shared memory.
		__shared__ float partialReductionOutputs[NUM_PARTIALS*NUM_CH];

		partialReductionOutputs[NUM_CH*partialIdx + 0] = outputL;
		partialReductionOutputs[NUM_CH*partialIdx + 1] = outputR;
		unsigned numActiveThreads = NUM_PARTIALS / 2;
		while (numActiveThreads > 0) {
			__syncthreads();
			if (partialIdx < numActiveThreads) {
				partialReductionOutputs[NUM_CH*partialIdx + 0] += partialReductionOutputs[NUM_CH*partialIdx + numActiveThreads*NUM_CH + 0];
				partialReductionOutputs[NUM_CH*partialIdx + 1] += partialReductionOutputs[NUM_CH*partialIdx + numActiveThreads*NUM_CH + 1];
			}
			numActiveThreads /= 2;
		}
		if (partialIdx == 0) {
			voiceState->sampleBuffer[bufferIdx + 0] = partialReductionOutputs[0];
			voiceState->sampleBuffer[bufferIdx + 1] = partialReductionOutputs[1];
		}
#else
		//host code
		//Since everything's computed iteratively, we can just add our outputs directly to the buffer.
		//First write to this sample must zero-initialize the buffer (not required in the GPU code).
		if (partialIdx == 0) {
			voiceState->sampleBuffer[bufferIdx + 0] = 0;
			voiceState->sampleBuffer[bufferIdx + 1] = 0;
		}
		voiceState->sampleBuffer[bufferIdx + 0] += outputL;
		voiceState->sampleBuffer[bufferIdx + 1] += outputR;
#endif
	}

	// called at the end of the block.
	// if parameterInfo.start != parameterInfo.end, then we copy the end parameters of this block to the start parameters for the next block.
	// this needs to be called for each sine wave.
	__device__ __host__ void updateVoiceParametersIfNeeded(SynthVoiceState *voiceState, unsigned voiceNum, unsigned partialIdx) {
		/*int transferSize = 16;
		int totalBytesToCopy = sizeof(ParameterStates);
		int numTransfers = (totalBytesToCopy + transferSize - 1) / transferSize;
		int numTransfersPerThread = (numTransfers + NUM_PARTIALS - 1) / NUM_PARTIALS;*/
		if (partialIdx == NUM_PARTIALS-1) {
			// TODO: only do this copy if the parameters have changed
			memcpy(&voiceState->parameterInfo.start, &voiceState->parameterInfo.end, sizeof(ParameterStates));
		}

	}

	// compute the output for ONE sine wave over the current block
	__device__ __host__ void computePartialOutput(SynthState *synthState, unsigned voiceNum, unsigned baseIdx, unsigned partialIdx, float fundamentalFreq, bool released) {
		SynthVoiceState *voiceState = &synthState->voiceStates[voiceNum];
		PartialState* myState = &voiceState->partialStates[partialIdx];
		myState->atBlockStart(voiceState, partialIdx, fundamentalFreq, released);
		// Get the base partial level (the hand-drawn frequency weights)
		float level = (1.0 / NUM_PARTIALS) * voiceState->parameterInfo.start.partialLevels[partialIdx];
		//printf("partialIdx: %i\n", partialIdx);
		for (int sampleIdx = 0; sampleIdx < BUFFER_BLOCK_SIZE; ++sampleIdx) {
			// Extract the sinusoidal portion of the wave.
			// float sinusoid = myState->sinusoid.next().imag();
			float sinusoid = myState->sinusoid.valueAtIdx(sampleIdx);
			// Get the ADSR/LFO volume envelope
			//float envelope = myState->volumeEnvelope.nextAsProduct();
			//float pan = myState->stereoPanEnvelope.nextAsSum();
			float envelope = myState->volumeEnvelope.productAtIdx(sampleIdx);
			float pan = myState->stereoPanEnvelope.sumAtIdx(sampleIdx);
			float unpanned = level*envelope*sinusoid;
			//float outputL = unpanned;
			//float outputR = unpanned;
			// full left = -1 pan. full right = +1 pan.
			// Use circular panning, where L^2 + R^2 = 1.0
			//   R(+1.0 pan) = 1.0, L(-1.0 pan) = 0.0, R(0.0 pan) = sqrt(1/2)
			//   L(+1.0 pan) = 0.0, L(-1.0 pan) = 1.0, L(0.0 pan) = sqrt(1/2)
			//   then R(pan) = sqrt((1+pan)/2)
			//   L(pan) = sqrt((1-pan)/2)
			// Note that L(pan)^2 + R(pan)^2 = 1.0, so energy is constant.
			// Must deal with pan values of magnitude > 1.0
			// Note the analog between sinusoidals:
			// sin(x)^2 + cos(x)^2 = 1.0 = L(pan)^2 + R(pan)^2
			// sin(pi/4) = cos(pi/4) = L(0.0) = R(0.0) = sqrt(1/2)
			// cos(0.0) = L(-1.0) = 1.0
			// cos(pi/2) = L(1.0) = 0.0
			// sin(0.0) = R(-1.0) = 0.0
			// sin(pi/2) = R(1.0) = 1.0
			// So, L(pan) = cos(pi/4 + pi/4*pan) = cos(pi/4*(1+pan))
			//     R(pan) = sin(pi/4 + pi/4*pan) = sin(pi/4*(1+pan))
			float angle = PI / 4 * (1 + pan);
			float outputL = unpanned * cosf(angle);
			float outputR = unpanned * sinf(angle);
			//float outputL = unpanned * sqrt(0.5*(1-pan));
			//float outputR = unpanned * sqrt(0.5*(1+pan));
			//linear pan implementation:
			//float outputL = unpanned * 0.5*(1 - pan);
			//float outputR = unpanned * 0.5*(1 + pan);

			reduceOutputs(voiceState, partialIdx, baseIdx + sampleIdx, outputL, outputR);
		}
		updateVoiceParametersIfNeeded(voiceState, voiceNum, partialIdx);
		// TODO: use a proper reduction algorithm to determine when the note is complete
		if (partialIdx == NUM_PARTIALS-1 && !myState->volumeEnvelope.isActiveAtEndOfBlock()) {
			// signal no more samples
			unsigned bufferEndIdx = NUM_CH * ((baseIdx + (BUFFER_BLOCK_SIZE - 1)) % CIRCULAR_BUFFER_LEN);
			voiceState->sampleBuffer[bufferEndIdx] = NAN;
		}
	}

	__global__ void evaluateSynthVoiceBlockKernel(SynthState *synthState, unsigned voiceNum, unsigned baseIdx, float fundamentalFreq, bool released) {
		int partialNum = threadIdx.x;
		computePartialOutput(synthState, voiceNum, baseIdx, partialNum, fundamentalFreq, released);
	}

	__host__ void evaluateSynthVoiceBlockOnCpu(float bufferB[BUFFER_BLOCK_SIZE*NUM_CH], unsigned voiceNum, unsigned sampleIdx, float fundamentalFreq, bool released) {
		// need to obtain a lock on the synth state
		std::unique_lock<std::mutex> stateLock(synthStateMutex);
		// move pointer to d_synthState into a local for easy debugging
		SynthState *synthState = d_synthState;
		for (int partialIdx = 0; partialIdx < NUM_PARTIALS; ++partialIdx) {
			computePartialOutput(synthState, voiceNum, sampleIdx, partialIdx, fundamentalFreq, released);
		}
		unsigned bufferStartIdx = NUM_CH * (sampleIdx % CIRCULAR_BUFFER_LEN);
		memcpy(bufferB, &synthState->voiceStates[voiceNum].sampleBuffer[bufferStartIdx], BUFFER_BLOCK_SIZE*NUM_CH*sizeof(float));
	}

	__host__ void evaluateSynthVoiceBlockCuda(float bufferB[BUFFER_BLOCK_SIZE*NUM_CH], unsigned voiceNum, unsigned sampleIdx, float fundamentalFreq, bool released) {
		evaluateSynthVoiceBlockKernel << <1, NUM_PARTIALS >> >(d_synthState, voiceNum, sampleIdx, fundamentalFreq, released);

		checkCudaError(hipGetLastError()); //check if error in kernel launch
		checkCudaError(hipDeviceSynchronize()); //check for error INSIDE the kernel

		//copy memory into the cpu buffer
		//Note: this will wait for the kernel to complete first.
		unsigned bufferStartIdx = NUM_CH * (sampleIdx % CIRCULAR_BUFFER_LEN);
		checkCudaError(hipMemcpy(bufferB, &d_synthState->voiceStates[voiceNum].sampleBuffer[bufferStartIdx], BUFFER_BLOCK_SIZE*NUM_CH*sizeof(float), hipMemcpyDeviceToHost));
	}

	void evaluateSynthVoiceBlock(float *bufferB, unsigned voiceNum, unsigned baseIdx, float fundamentalFreq, bool released) {
		doStartupOnce();
		if (hasCudaDevice()) {
			evaluateSynthVoiceBlockCuda(bufferB, voiceNum, baseIdx, fundamentalFreq, released);
		} else {
			evaluateSynthVoiceBlockOnCpu(bufferB, voiceNum, baseIdx, fundamentalFreq, released);
		}
	}

	static void memcpyHostToSynthState(void *dest, const void *src, std::size_t numBytes) {
		// if running on device, copy params to GPU via hipMemcpy, else normal memcpy on cpu.
		if (hasCudaDevice()) {
			// hipMemcpy is synchronous, so concurrency is dealt with automatically
			checkCudaError(hipMemcpy(dest, src, numBytes, hipMemcpyHostToDevice));
		} else {
			// else, copy them using normal memcpy
			// Must first obtain a lock to the synth data.
			std::unique_lock<std::mutex> stateLock(synthStateMutex);
			memcpy(dest, src, numBytes);
		}
	}

	static void memsetSynthState(void *dest, int value, std::size_t numBytes) {
		// if running on device, use hipMemset, else normal memset
		if (hasCudaDevice()) {
			// hipMemset is synchronous, so concurrency is dealt with automatically
			checkCudaError(hipMemset(dest, value, numBytes));
		} else {
			// else, use normal memset
			// Must first obtain a lock to the synth data.
			std::unique_lock<std::mutex> stateLock(synthStateMutex);
			memset(dest, value, numBytes);
		}
	}

	static void copyParameterStates(const ParameterStates *newParameters, ParameterStates *dest) {
		memcpyHostToSynthState(dest, newParameters, sizeof(ParameterStates));
	}

	void parameterStatesChanged(const ParameterStates *newParameters) {
		doStartupOnce();
		static bool hasInitStartParams = false;

		for (int i = 0; i < MAX_SIMULTANEOUS_SYNTH_NOTES; ++i) {
			// If this is the first time we've received parameter states, then that means parameterInfo.start is uninitialized.
			if (!hasInitStartParams) {
				copyParameterStates(newParameters, &d_synthState->voiceStates[i].parameterInfo.start);
			}
			copyParameterStates(newParameters, &d_synthState->voiceStates[i].parameterInfo.end);
		}
		hasInitStartParams = true;
	}

	void onNoteStart(unsigned voiceNum) {
		doStartupOnce();
		// need to go through and properly initialize all the note's state information:
		//   partial phases, ADSR states, etc.
		PartialState partialStates[NUM_PARTIALS];
		for (int i = 0; i < NUM_PARTIALS; ++i) {
			partialStates[i] = PartialState(d_synthState, voiceNum, i);
		}
		memcpyHostToSynthState(&d_synthState->voiceStates[voiceNum].partialStates, partialStates, sizeof(PartialState)*NUM_PARTIALS);
		memsetSynthState(&d_synthState->voiceStates[voiceNum].sampleBuffer, 0, CIRCULAR_BUFFER_LEN*NUM_CH*sizeof(float));
	}

}