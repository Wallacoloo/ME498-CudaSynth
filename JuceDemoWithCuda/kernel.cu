#include "hip/hip_runtime.h"
#include "kernel.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <string.h> //for memset
#include <assert.h>
#include <stdlib.h> //for atexit
#include <mutex>
#include <thread> //for unique_lock

#include "defines.h"

#define CIRCULAR_BUFFER_LEN MAX_DELAY_EFFECT_LENGTH

namespace kernel {

	// Contains info about the parameter states at ANY sample in the block
	struct FullBlockParameterInfo {
		ParameterStates start;
		ParameterStates end;
	};

	// Packages all the state-related information for the synth in one class to store persistently on the device
	struct SynthState {
		FullBlockParameterInfo parameterInfo;
		float sampleBuffer[CIRCULAR_BUFFER_LEN*NUM_CH];
	};

	// When summing the outputs at a specific frame for each partial, we use a reduction method.
	// This reduction method requires a temporary array in shared memory.
	__shared__ float partialReductionOutputs[NUM_PARTIALS*NUM_CH];

	// this is a circular buffer of sample data (interleaved by channel number) stored on the device
	// It is persistent and lengthy, in order to accomodate the delay effect.
	SynthState *d_synthState = NULL;

	// When running on the cpu, we need to control concurrent access to the synth state
	std::mutex synthStateMutex;


	void checkCudaError(hipError_t e) {
		if (e != hipSuccess) {
			printf("Cuda Error: %s\n", hipGetErrorString(e));
			printf("Aborting\n");
			exit(1);
		}
	}

	bool _hasCudaDevice() {
		int deviceCount;
		hipError_t err = hipGetDeviceCount(&deviceCount);
		// if we get a cuda error, it may be because the system has no cuda dlls.
		bool useCuda = (err == hipSuccess && deviceCount != 0);
		printf("Using Cuda? %i\n", useCuda);
		return useCuda;
	}

	bool hasCudaDevice() {
		//only check for the presence of a device once.
		static bool hasDevice = _hasCudaDevice();
		return hasDevice;
	}

	// code to run at shutdown (free buffers, etc)
	void teardown() {
		// free the sample buffer if we allocated it and it hasn't already been freed.
		if (d_synthState != NULL) {
			if (hasCudaDevice()) {
				checkCudaError(hipFree(d_synthState));
			}
			else {
				free(d_synthState);
			}
			// avoid double-frees
			d_synthState = NULL;
		}
	}

	// code to run on first-time audio calculation
	void startup() {
		atexit(&teardown);
		if (hasCudaDevice()) {
			// allocate sample buffer on device
			checkCudaError(hipMalloc(&d_synthState, sizeof(SynthState)));
		}
		else {
			// allocate sample buffer on cpu
			d_synthState = (SynthState*)malloc(sizeof(SynthState));
		}
	}

	void doStartupOnce() {
		static bool hasInit = false;
		if (!hasInit) {
			startup();
			hasInit = true;
		}
	}


	__device__ __host__ void reduceOutputs(SynthState *synthState, unsigned partialIdx, int sampleIdx, float outputL, float outputR) {
		//algorithm: given 8 outputs, [0, 1, 2, 3, 4, 5, 6, 7]
		//first iteration: 4 active threads. 
		//  Thread 0 adds i0 to i(0+4). Thread 1 adds i1 to i(1+4). Thread 2 adds i2 to i(2+4). Thread 3 adds i3 to i(3+4)
		//  Output now: [4, 6, 8, 10,   4, 5, 6, 7]
		//second iteration: 2 active threads.
		//  Thread 0 adds i0 to i(0+2). Thread 1 adds i1 to i(1+2)
		//  Output now: [12, 16,   8, 10, 4, 5, 6, 7]
		//third iteration: 1 active thread.
		//  Thread 0 adds i0 to i(0+1).
		//  Output now: [28,   16, 8, 10, 4, 5, 6, 7]
		//fourth iteration: 0 active threads -> exit
		unsigned bufferIdx = NUM_CH * (sampleIdx % CIRCULAR_BUFFER_LEN);
#ifdef __CUDA_ARCH__
		//device code
		partialReductionOutputs[NUM_CH*partialIdx + 0] = outputL;
		partialReductionOutputs[NUM_CH*partialIdx + 1] = outputR;
		unsigned numActiveThreads = NUM_PARTIALS / 2;
		while (numActiveThreads > 0) {
			__syncthreads();
			partialReductionOutputs[NUM_CH*partialIdx + 0] += partialReductionOutputs[NUM_CH*partialIdx + numActiveThreads*NUM_CH + 0];
			partialReductionOutputs[NUM_CH*partialIdx + 1] += partialReductionOutputs[NUM_CH*partialIdx + numActiveThreads*NUM_CH + 1];
			numActiveThreads /= 2;
		}
		if (partialIdx == 0) {
			synthState->sampleBuffer[bufferIdx + 0] = partialReductionOutputs[0];
			synthState->sampleBuffer[bufferIdx + 1] = partialReductionOutputs[1];
		}
#else
		//host code
		//Since everything's computed iteratively, we can just add our outputs directly to the buffer.
		//First write to this sample must zero-initialize the buffer (not required in the GPU code).
		if (partialIdx == 0) {
			synthState->sampleBuffer[bufferIdx + 0] = 0;
			synthState->sampleBuffer[bufferIdx + 1] = 0;
		}
		synthState->sampleBuffer[bufferIdx + 0] += outputL;
		synthState->sampleBuffer[bufferIdx + 1] += outputR;
#endif
	}


	__device__ __host__ void computePartialOutput(SynthState *synthState, unsigned baseIdx, unsigned partialIdx, float fundamentalFreq) {
		float angleDelta = fundamentalFreq * INV_SAMPLE_RATE * (partialIdx + 1);
		for (int sampleIdx = 0; sampleIdx < BUFFER_BLOCK_SIZE; ++sampleIdx) {
			float outputL, outputR;
			outputL = outputR = (1.0 / NUM_PARTIALS)*sinf((baseIdx + sampleIdx) * angleDelta);
			reduceOutputs(synthState, partialIdx, baseIdx + sampleIdx, outputL, outputR);
		}
	}

	__global__ void evaluateSynthVoiceBlockKernel(SynthState *synthState, unsigned baseIdx, float fundamentalFreq) {
		int partialNum = threadIdx.x;
		computePartialOutput(synthState, baseIdx, partialNum, fundamentalFreq);
	}

	__host__ void evaluateSynthVoiceBlockOnCpu(float *bufferB, unsigned sampleIdx, float fundamentalFreq) {
		// need to obtain a lock on the synth state
		std::unique_lock<std::mutex> stateLock(synthStateMutex);
		for (int partialIdx = 0; partialIdx < NUM_PARTIALS; ++partialIdx) {
			computePartialOutput(d_synthState, sampleIdx, partialIdx, fundamentalFreq);
		}
		unsigned bufferStartIdx = NUM_CH * (sampleIdx % CIRCULAR_BUFFER_LEN);
		memcpy(bufferB, &d_synthState->sampleBuffer[bufferStartIdx], BUFFER_BLOCK_SIZE*NUM_CH*sizeof(float));
	}

	__host__ void evaluateSynthVoiceBlockCuda(float *bufferB, unsigned sampleIdx, float fundamentalFreq) {
		// update the ending parameter states of this block
		// if (newParameters) {
		//	checkCudaError(hipMemcpy(&d_synthState->parameterInfo.end, newParameters, sizeof(ParameterStates), hipMemcpyHostToDevice));
		//}
		evaluateSynthVoiceBlockKernel << <1, NUM_PARTIALS >> >(d_synthState, sampleIdx, fundamentalFreq);

		checkCudaError(hipGetLastError()); //check if error in kernel launch
		checkCudaError(hipDeviceSynchronize()); //check for error INSIDE the kernel

		//copy memory into the cpu buffer
		//Note: this will wait for the kernel to complete first.
		unsigned bufferStartIdx = NUM_CH * (sampleIdx % CIRCULAR_BUFFER_LEN);
		checkCudaError(hipMemcpy(bufferB, &d_synthState->sampleBuffer[bufferStartIdx], BUFFER_BLOCK_SIZE*NUM_CH*sizeof(float), hipMemcpyDeviceToHost));
	}

	void evaluateSynthVoiceBlock(float *bufferB, unsigned baseIdx, float fundamentalFreq) {
		doStartupOnce();
		if (hasCudaDevice()) {
			evaluateSynthVoiceBlockCuda(bufferB, baseIdx, fundamentalFreq);
		} else {
			evaluateSynthVoiceBlockOnCpu(bufferB, baseIdx, fundamentalFreq);
		}
	}

	void parameterStatesChanged(const ParameterStates *newParameters) {
		// if running on device, copy params to GPU (hipMemcpy
		if (hasCudaDevice()) {
			// hipMemcpy is synchronous, so concurrency is dealt with automatically
			checkCudaError(hipMemcpy(&d_synthState->parameterInfo.end, newParameters, sizeof(ParameterStates), hipMemcpyHostToDevice));
		} else {
			// else, copy them using normal memcpy
			// Must first obtain a lock to the synth data.
			std::unique_lock<std::mutex> stateLock(synthStateMutex);
			memcpy(&d_synthState->parameterInfo.end, newParameters, sizeof(ParameterStates));
		}
	}

}