#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

#include "defines.h"

__global__ void fillSineWaveKernel(float *buffer, unsigned baseIdx, float level, float angleDelta) {
	int bufIdx = threadIdx.x;
	buffer[bufIdx] = level*sin((baseIdx + bufIdx) * angleDelta);
}

void cudaFillSineWaveVoice(float *bufferB, unsigned baseIdx, float level, float angleDelta) {
	float *gpuOutBuff;
	hipMalloc(&gpuOutBuff, BUFFER_BLOCK_SIZE*sizeof(float));
	fillSineWaveKernel<<<1, BUFFER_BLOCK_SIZE >>>(gpuOutBuff, baseIdx, level, angleDelta);
	//copy memory into the cpu buffer
	hipMemcpy(bufferB, gpuOutBuff, BUFFER_BLOCK_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(gpuOutBuff);
}
